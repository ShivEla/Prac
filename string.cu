#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_WORDS 1000
#define MAX_WORD_LENGTH 50

__global__ void wordCountKernel(char *str, int *histogram, int numWords) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid < numWords) {
        atomicAdd(&histogram[str[tid]], 1);
        tid += stride;
    }
}

int main() {
    char input[] = "Hello world hello CUDA hello world world";
    char *dev_input;
    int *histogram, *dev_histogram;

    // Allocate memory on host
    int numWords = 0;
    char *words[MAX_WORDS];
    char *token = strtok(input, " ");
    while (token != NULL && numWords < MAX_WORDS) {
        words[numWords++] = token;
        token = strtok(NULL, " ");
    }

    // Allocate memory on device
    hipMalloc(&dev_input, strlen(input) + 1);
    hipMalloc(&dev_histogram, MAX_WORDS * sizeof(int));

    // Copy input string to device
    hipMemcpy(dev_input, input, strlen(input) + 1, hipMemcpyHostToDevice);

    // Initialize histogram to zeros
    hipMemset(dev_histogram, 0, MAX_WORDS * sizeof(int));

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (numWords + blockSize - 1) / blockSize;
    wordCountKernel<<<numBlocks, blockSize>>>(dev_input, dev_histogram, numWords);

    // Copy histogram back to host
    histogram = (int *)malloc(MAX_WORDS * sizeof(int));
    hipMemcpy(histogram, dev_histogram, MAX_WORDS * sizeof(int), hipMemcpyDeviceToHost);

    // Print word counts
    for (int i = 0; i < numWords; ++i) {
        printf("%s: %d\n", words[i], histogram[i]);
    }

    // Cleanup
    free(histogram);
    hipFree(dev_input);
    hipFree(dev_histogram);

    return 0;
}


#Q1.PRINTING THREAD IDS
#include <stdio.h>

_global_ void printThreadID()
{
    int tid_x = threadIdx.x; // Thread index in x dimension
    int tid_y = threadIdx.y; // Thread index in y dimension
    int tid = tid_x + tid_y * blockDim.x; // Global thread ID

    printf("Thread ID: %d, threadIdx.x: %d, threadIdx.y: %d\n", tid, tid_x, tid_y);
}

int main()
{
    // Define grid and block dimensions
    dim3 threadsPerBlock(2, 3); // 2 threads in x dimension, 3 threads in y dimension
    dim3 numBlocks(1, 1); // 1 block in x dimension, 1 block in y dimension

    // Launch kernel
    printThreadID<<<numBlocks, threadsPerBlock>>>();

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    return 0;
}
#-------------------------------------------------------------------
Q2.SUM OF ELEMENTS IN 2D ARRAY
#include <stdio.h>

#define N 4

_global_ void sum2DArray(int *array, int *result)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    atomicAdd(result, array[tid]);
}

int main()
{
    int h_array[N][N] = {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
    };
    int h_result = 0;

    int *d_array, *d_result;
    hipMalloc((void **)&d_array, N * N * sizeof(int));
    hipMalloc((void **)&d_result, sizeof(int));

    hipMemcpy(d_array, h_array, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    sum2DArray<<<1, threadsPerBlock>>>(d_array, d_result);

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum of elements in 2D array: %d\n", h_result);

    hipFree(d_array);
    hipFree(d_result);

    return 0;
}
#------------------------------------------------------------------
Q3.CALCULATE DISTANCE OF ALL THE POINTS IN A GRID TO A SPECIFIC POINT (X,Y) WITH SINGLE BLOCK AND MULTIPLE THREADS

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 8

// Memory Allocated in Device
_device_ float dgrid[N][N];

// Kernel Function
_global_ void findDistance(int x, int y)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    float n = ((i - x) * (i - x)) + ((j - y) * (j - y));
    dgrid[i][j] = sqrt(n);
}

// Main Function
void main()
{
    int i, j;

    // Memory Allocated in Host
    float hgrid[N][N];

    // 1D Grid
    // 2D Block
    dim3 dBlock(N, N);

    // ----
    printf("Enter the x coordinate of node : ");
    scanf_s("%d", &i);

    printf("Enter the y coordinate of node : ");
    scanf_s("%d", &j);

    // Calling the kernel function with 1 - Grid, 1 - 2D_Block, 16x16 - Threads
    findDistance<<<1, dBlock>>>(i, j);
    // ----

    // Copy the matrix from device to host to print to console
    hipMemcpyFromSymbol(&hgrid, HIP_SYMBOL(dgrid), sizeof(dgrid));

    printf("Values in hgrid!\n\n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
            printf("\t%.0lf", hgrid[i][j]);
        printf("\n\n");
    }
}
#-------------------------------------------------------------------
Q4.CALCULATE DISTANCE OF ALL THE POINTS IN A GRID TO A SPECIFIC POINT (X,Y) WITH MULTIPLE BLOCK AND MULTIPLE THREADS

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 16
#define D 2

// Memory Allocated in Device
_device_ float dgrid[N * D][N * D];

// Kernel Function
_global_ void findDistance(int x, int y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float n = ((i - x) * (i - x)) + ((j - y) * (j - y));
    dgrid[i][j] = sqrt(n);
}

// Main Function
void main()
{
    int i, j;

    // Memory Allocated in Host
    float hgrid[N * D][N * D];

    // 2D Grid (4 * 4 Blocks)
    dim3 dGrid(D, D);

    // 2D Block (16 * 16)
    dim3 dBlock(N, N);

    printf("Enter the x coordinate of node : ");
    scanf_s("%d", &i);
    printf("Enter the y coordinate of node : ");
    scanf_s("%d", &j);

    // Calling the kernel function with 1 - 2D_Grid, 1 - 2D_Block, 16x16 - Threads
    findDistance<<<dGrid, dBlock>>>(i, j);

    // Copy the matrix from device to host to print to console
    hipMemcpyFromSymbol(&hgrid, HIP_SYMBOL(dgrid), sizeof(dgrid));

    printf("Values in hgrid!\n\n");
    for (i = 0; i < N * D; i++)
    {
        for (j = 0; j < N * D; j++)
            printf("\t%.0lf", hgrid[i][j]);
        printf("\n\n");
    }
}
#----------------------------------------------------------------
Q5.CHARACTER ARRAY COPYING
#include <stdio.h>

#define N 10

_global_ void copyCharArrays(char *src, char *dest)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    dest[tid] = src[tid];
}

int main()
{
    char h_src[N] = "HelloCUDA";
    char h_dest[N];

    char *d_src, *d_dest;
    hipMalloc((void **)&d_src, N * sizeof(char));
    hipMalloc((void **)&d_dest, N * sizeof(char));

    hipMemcpy(d_src, h_src, N * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 4; // Threads per block
    int numBlocks = (N + blockSize - 1) / blockSize; // Calculate number of blocks needed

    copyCharArrays<<<numBlocks, blockSize>>>(d_src, d_dest);

    hipMemcpy(h_dest, d_dest, N * sizeof(char), hipMemcpyDeviceToHost);

    printf("Copied String: %s\n", h_dest);

    hipFree(d_src);
    hipFree(d_dest);

    return 0;
}
#-------------------------------------------------------------------
#Q6.CHARACTER ARRAY MANIPULATION
#include <stdio.h>

#define N 10

_global_ void manipulateCharArray(char *array)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (array[tid] >= 'a' && array[tid] <= 'z')
        array[tid] -= 32; // Convert lowercase letter to uppercase
}

int main()
{
    char h_array[N] = "helloCUDA";

    char *d_array;
    hipMalloc((void **)&d_array, N * sizeof(char));

    hipMemcpy(d_array, h_array, N * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 4; // Threads per block
    int numBlocks = (N + blockSize - 1) / blockSize; // Calculate number of blocks needed

    manipulateCharArray<<<numBlocks, blockSize>>>(d_array);

    hipMemcpy(h_array, d_array, N * sizeof(char), hipMemcpyDeviceToHost);

    printf("Manipulated String: %s\n", h_array);

    hipFree(d_array);

    return 0;
}
#--------------------------------------------------------------------
Q7.WORD COUNTER
%%cuda
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_LENGTH 100
#define MAX_WORDS 100

_device_ int myStrCmp(const char *str1, const char *str2)
{
    int i = 0;
    while (str1[i] != '\0' && str2[i] != '\0' && str1[i] == str2[i])
    {
        i++;
    }
    return (str1[i] - str2[i]);
}

_device_ void myStrCpy(char *dest, const char *src)
{
    int i = 0;
    while ((dest[i] = src[i]) != '\0')
    {
        i++;
    }
}

// Kernel function to count word frequency
_global_ void countWordFrequency(char *sentence, int *wordCounts, int numWords, char *uniqueWords, int *totalWords)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numWords && wordCounts[tid] == 0)
    {
        int count = 1; // Initialize count for current word
        for (int i = tid + 1; i < numWords; ++i)
        {
            if (myStrCmp(&sentence[tid * MAX_LENGTH], &sentence[i * MAX_LENGTH]) == 0)
            {
                count++; // Increment count if the word matches
                wordCounts[i] = 1; // Mark word as counted
            }
        }
        wordCounts[tid] = count;

        // Store unique words
        if (wordCounts[tid] == 1)
        {
            int index = atomicAdd(totalWords, 1);
            myStrCpy(&uniqueWords[index * MAX_LENGTH], &sentence[tid * MAX_LENGTH]);
        }
    }
}

int main()
{
    char h_sentence[MAX_WORDS][MAX_LENGTH] = {
        "hello", "world", "hello", "cuda", "world"
    };
    int numWords = 5;

    // Copy sentence to device memory
    char *d_sentence;
    hipMalloc((void **)&d_sentence, numWords * MAX_LENGTH * sizeof(char));
    hipMemcpy(d_sentence, h_sentence, numWords * MAX_LENGTH * sizeof(char), hipMemcpyHostToDevice);

    // Allocate memory for word counts on device
    int *d_wordCounts;
    hipMalloc((void **)&d_wordCounts, numWords * sizeof(int));

    // Allocate memory for unique words on device
    char *d_uniqueWords;
    hipMalloc((void **)&d_uniqueWords, MAX_WORDS * MAX_LENGTH * sizeof(char));

    // Total number of words
    int *d_totalWords;
    hipMalloc((void **)&d_totalWords, sizeof(int));
    hipMemcpy(d_totalWords, &numWords, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to count word frequency and extract unique words
    int blockSize = 256;
    int numBlocks = (numWords + blockSize - 1) / blockSize;
    countWordFrequency<<<numBlocks, blockSize>>>(d_sentence, d_wordCounts, numWords, d_uniqueWords, d_totalWords);

    // Copy unique words back to host
    char h_uniqueWords[MAX_WORDS][MAX_LENGTH];
    int totalWords;
    hipMemcpy(&totalWords, d_totalWords, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_uniqueWords, d_uniqueWords, totalWords * MAX_LENGTH * sizeof(char), hipMemcpyDeviceToHost);

    // Print word counts for unique words
    printf("\nWord Frequencies:\n");
    for (int i = 0; i < totalWords; ++i)
    {
        int count = 0;
        for (int j = 0; j < numWords; ++j)
        {
            if (strcmp(h_uniqueWords[i], h_sentence[j]) == 0)
            {
                count++;
            }
        }
        if (count != 0) {
            printf("%s : %d\n", h_uniqueWords[i], count);
        }
    }

    // Free device memory
    hipFree(d_sentence);
    hipFree(d_wordCounts);
    hipFree(d_uniqueWords);
    hipFree(d_totalWords);

    return 0;
}
